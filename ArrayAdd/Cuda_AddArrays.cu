
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <iostream>
__global__ void mykernel(int *a, int *b, int *c, int n)
{
  int index = blockIdx.x*blockDim.x + threadIdx.x;

  if (index < n)
  {
    c[index] = a[index] + b[index];

  }
}
int* genVector(int *p, int n)
{
  std::cout << " Vector : " ;
  for (int i = 0; i < n; i++)
  {
    p[i] = rand()/100;
    std::cout << p[i] << " ";
  }
  std::cout << "" << std::endl;
  return p;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}
int main(void)
{
  int *h_a, *h_b, *h_c;
  int *d_a, *d_b, *d_c;
  
  int n = 16;
  int NUM_THREADS = 16;
  int NUM_BLOCKS = (int)ceil(n + NUM_THREADS+1)/NUM_THREADS;
 
  std::size_t bytes = sizeof(int)*n;

  h_a = (int*)malloc(bytes);
  h_b = (int*)malloc(bytes);
  h_c = (int*)malloc(bytes);
 
  gpuErrchk(hipMalloc(&d_a, bytes));
  gpuErrchk(hipMalloc(&d_b, bytes));
  gpuErrchk(hipMalloc(&d_c, bytes));
  
  genVector(h_a, n);
  genVector(h_b, n);

  gpuErrchk(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice));
  hipMemcpy(d_c, h_c, bytes, hipMemcpyHostToDevice);

 

  mykernel <<<NUM_BLOCKS, NUM_THREADS >>>(d_a, d_b, d_c, n);
 
  gpuErrchk(hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost));
 

  std::cout << "result : ";
  for (int i = 0; i < n; i++)
    std::cout << (h_c[i]) << " ";
  
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  
  free(h_a);
  free(h_b);
  free(h_c);
}